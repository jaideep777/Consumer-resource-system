#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <cstdlib>
#include <string>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#include "../headers/consumers.h"
//#include "../headers/graphics.h"
#include "../utils/cuda_vector_math.cuh"
#include "../utils/cuda_device.h"


// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// KERNEL to set up RANDOM GENERATOR STATES
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

__global__ void csRngStateSetup_kernel(int * rng_Seeds, hiprandState * rngStates, int nc){
	int tid_u = threadIdx.x + blockIdx.x*blockDim.x;	// each block produces different random numbers
	if (tid_u >= nc) return;
	
	hiprand_init (rng_Seeds[tid_u], 0, 0, &rngStates[tid_u]);
}

#define CS_PP_SEED 777 //time(NULL)

void ConsumerSystem::initRNG(){
	srand(CS_PP_SEED);
	for (int i=0; i<nc; ++i) cs_seeds_h[i] = rand(); 
	hipMemcpy( cs_seeds_dev, cs_seeds_h, sizeof(int)*nc, hipMemcpyHostToDevice);

	int nt = min(256, nc), nb = (nc-1)/nt+1;
	csRngStateSetup_kernel <<< nb, nt>>> (cs_seeds_dev, cs_dev_XWstates, nc);
	getLastCudaError("RNG_kernel_launch");
}


// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//          CONSUMER SYSTEM
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

void ConsumerSystem::init(Initializer &I){
	nx = I.getScalar("nx"); 
	ny = I.getScalar("ny");
	L  = I.getScalar("L");
	nc = I.getScalar("nc");
	dL = L/nx;
	
	ke_lmax = I.getScalar("Ke_cutoff");		// bound for exploitation kernel in length units
	ke_nmax = int(ke_lmax/(L/nx));
	ke_sd = I.getScalar("Ke_sd");
	cout << "ke_nmax = " << ke_nmax << endl;

	getLastCudaError("resgrid");
	
	// allocate space
	int ke_arrlen = (2*ke_nmax+1)*(2*ke_nmax+1);  // ke goes from [-ke_nmax, ke_nmax]
	ke = new float[ke_arrlen];
	for (int j=-ke_nmax; j<=ke_nmax; ++j){
		for (int i=-ke_nmax; i<=ke_nmax; ++i){
			float ker = 1-(i*i+j*j)*dL*dL/ke_sd/ke_sd;
			if (ker < 0) ker = 0;
			ke[ix2(i+ke_nmax, j+ke_nmax, 2*ke_nmax+1)] = ker;
		}
	}
	hipMalloc((void**)&ke_dev, sizeof(float)*ke_arrlen);
	hipMemcpy(ke_dev, ke, ke_arrlen*sizeof(float), hipMemcpyHostToDevice);
	
	ke_all = new float[nx*ny];
	for (int i=0; i<nx*ny; ++i) ke_all[i] = 0;
	hipMalloc((void**)&ke_all_dev, sizeof(float)*nx*ny);

	consumers.resize(nc);
	for (int i=0; i<nc; ++i){
		consumers[i].pos   = runif2(0,L,0,L);
		consumers[i].pos_i = pos2cell(consumers[i].pos, dL);
		consumers[i].RT    = I.getScalar("RT0");
		consumers[i].Kdsd  = I.getScalar("kdsd0");
		consumers[i].h     = I.getScalar("h0");
		if (i<nc/2) consumers[i].h = 0.1;
		else 		consumers[i].h = 0.3;
		
		cout << consumers[i].pos.x << " " << consumers[i].pos.y << ", "   
			 << consumers[i].pos_i.x << " " << consumers[i].pos_i.y   << endl;

	}
	hipMalloc((void**)&h_dev, sizeof(float)*nc);
	hipMalloc((void**)&pos_i_dev, sizeof(int2)*nc);	
	hipMalloc((void**)&rc_dev, sizeof(float)*nc);
	hipMalloc((void**)&RT_dev, sizeof(float)*nc);
	hipMalloc((void**)&kdsd_dev, sizeof(float)*nc);
	
	hipMalloc((void**)&nd_dev, sizeof(float)*nc);
	hipMalloc((void**)&lenDisp_dev, sizeof(float)*nc);

	vc_Tw = I.getScalar("payoff_Tw");
	hipMalloc((void**)&vc_window_dev, sizeof(float)*nc*vc_Tw);
	hipMalloc((void**)&vc_dev, sizeof(float)*nc);
	float vc
	
	hipMemcpy2D((void*)h_dev, sizeof(float), (void*)&consumers[0].h, sizeof(Consumer), sizeof(float),  nc, hipMemcpyHostToDevice);	
	hipMemcpy2D((void*)RT_dev, sizeof(float), (void*)&consumers[0].RT, sizeof(Consumer), sizeof(float),  nc, hipMemcpyHostToDevice);	
	hipMemcpy2D((void*)kdsd_dev, sizeof(float), (void*)&consumers[0].Kdsd, sizeof(Consumer), sizeof(float),  nc, hipMemcpyHostToDevice);	
	hipMemcpy2D((void*)pos_i_dev, sizeof(int2), (void*)&consumers[0].pos_i, sizeof(Consumer), sizeof(int2),  nc, hipMemcpyHostToDevice);	
	getLastCudaError("memcpy2D");

	
	cs_seeds_h = new int[nc];
	hipMalloc((void**)&cs_seeds_dev, nc*sizeof(int));
	hipMalloc((void**)&cs_dev_XWstates, nc*sizeof(hiprandState));
	
	initRNG();
	
	// create Pointset shape to display consumers
	cons_shape = PointSet("res", false, nc, 0, L);	// the res shader is a generic shader for colormaps
	cons_shape.nVertices = nc;
	cons_shape.createShaders();
	float2 tmp[nc]; 
	for (int i=0; i<nc; ++i) {
		tmp[i] = cell2pos(consumers[i].pos_i, dL);
//		cout << consumers[i].pos_i.x << " " << consumers[i].pos_i.y  << ", " << tmp[i].x << " " << tmp[i].y << endl;
	}
	cons_shape.createVBO(tmp, cons_shape.nVertices*sizeof(float2));	
	cons_shape.createColorBuffer();
	cons_shape.setDefaultColor();
	cons_shape.palette = createPalette_ramp(nc, Colour_rgb(0,0.9,0), Colour_rgb(1,0,0));
	printPalette(cons_shape.palette);
}


void ConsumerSystem::graphics_updateArrays(){

	// positions buffer
	hipMemcpy2D((void*)&consumers[0].pos_i, sizeof(Consumer), (void*)pos_i_dev, sizeof(int2), sizeof(int2),  nc, hipMemcpyDeviceToHost);	
	float2 tmp[nc]; 
	for (int i=0; i<nc; ++i) {
		tmp[i] = cell2pos(consumers[i].pos_i, dL);
//		cout << consumers[i].pos_i.x << " " << consumers[i].pos_i.y  << ", " << tmp[i].x << " " << tmp[i].y << endl;
	}
	glBindBuffer(GL_ARRAY_BUFFER, cons_shape.vbo_ids[0]); 	// Bring 1st buffer into current openGL context
	glBufferData(GL_ARRAY_BUFFER, nc*sizeof(float2), tmp, GL_DYNAMIC_DRAW); 
	glBindBuffer(GL_ARRAY_BUFFER, 0); 	// Bring 1st buffer into current openGL context
	
	// color buffer
	float h_tmp[nc];
	for (int i=0; i<nc; ++i) {
		h_tmp[i] = consumers[i].h;
	}
	cons_shape.updateColors(h_tmp, nc);
	
}


__global__ void calc_exploitation_kernels_kernel(float* ke_all, int2* pos_cell, float* h, int nc, float* ke, int rkn, int nx){
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nc) return;

	int ixc = pos_cell[tid].x;
	int iyc = pos_cell[tid].y;
	float hc = h[tid];

	for (int i=-rkn; i<=rkn; ++i){
		for (int j=-rkn; j<=rkn; ++j){
			int iK = makePeriodicID(ixc+i, nx);
			int jK = makePeriodicID(iyc+j, nx);
			ke_all[ix2(iK, jK, nx)] += hc * ke[ix2(i+rkn, j+rkn, (2*rkn+1))];
		}
	}	
	
}


void::ConsumerSystem::updateExploitationKernels(){
	
	for (int i=0; i<nx*ny; ++i) ke_all[i] = 0;	// reset exploitation kernels on host
	hipMemcpy(ke_all_dev, ke_all, nx*ny*sizeof(float), hipMemcpyHostToDevice); // reset ke_all_dev to zeros
	int nt = min(256, nc); int nb = 1; 
	calc_exploitation_kernels_kernel <<< nb, nt >>> (ke_all_dev, pos_i_dev, h_dev, nc, ke_dev, ke_nmax, nx);
	getLastCudaError("exploitation kernel");

//	hipMemcpy(ke_all, ke_all_dev, nx*ny*sizeof(float), hipMemcpyDeviceToHost);
//	ofstream fout("ke_all.txt");
//	for (int j=0; j<ny; ++j){
//		for (int i=0; i<nx; ++i){
//			fout << ke_all[ix2(i,j,nx)] << "\t";
//		}
//		fout << "\n";
//	}
//	fout << "\n";
}



__global__ void calc_resource_consumed_kernel(float *res, float* rc_vec, int2* pos_cell, float* h, int nc, float* ke, int rkn, int nx){

	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nc) return;

	int ixc = pos_cell[tid].x;
	int iyc = pos_cell[tid].y;
	float hc = h[tid];

	float R_avail = 0;
	for (int i=-rkn; i<=rkn; ++i){
		for (int j=-rkn; j<=rkn; ++j){
			int iK = makePeriodicID(ixc+i, nx);
			int jK = makePeriodicID(iyc+j, nx);
			R_avail += res[ix2(iK, jK, nx)] * ke[ix2(i+rkn, j+rkn, (2*rkn+1))];
		}
	}	
	rc_vec[tid] = hc * R_avail;

}

void::ConsumerSystem::calcResConsumed(float * resource_dev){
	
	int nt = min(256, nc); int nb = 1; 
	calc_resource_consumed_kernel <<< nb, nt >>> (resource_dev, rc_dev, pos_i_dev, h_dev, nc, ke_dev, ke_nmax, nx);
	getLastCudaError("rc kernel");

//	hipMemcpy2D(&consumers[0].rc, sizeof(Consumer), rc_dev, sizeof(float), sizeof(float), nc, hipMemcpyDeviceToHost);
//	for (int i=0; i<nc; ++i){
//		cout << consumers[i].rc << "\n";
//	}
//	cout << "\n";
}


__global__ void disperse_kernel(float * res, int2 * pos_cell, 
								float * kdsd_vec, float * RT_vec, 
								hiprandState * RNG_states, 
								float L, int nc, int nx, 
								float * lenDisp, float * nd){
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nc) return;
	
	int ixc = pos_cell[tid].x;
	int iyc = pos_cell[tid].y;

	float p_disperse = 1/(1+exp(10*(res[ix2(ixc, iyc, nx)] - RT_vec[tid])));	
	float b_disperse = hiprand_uniform(&RNG_states[tid]) < p_disperse;
	
	float len   = fabs(hiprand_normal(&RNG_states[tid]))*kdsd_vec[tid];
	float theta = hiprand_uniform(&RNG_states[tid])*2*3.14159;
	
	float xdisp = b_disperse*len*cos(theta);
	float ydisp = b_disperse*len*sin(theta);
	
	float2 xnew = make_float2(pos_cell[tid].x + xdisp, 
							  pos_cell[tid].y + ydisp );
	makePeriodic(xnew.x, 0, L);
	makePeriodic(xnew.y, 0, L);
	
	pos_cell[tid] = pos2cell(xnew, L/nx);
	lenDisp[tid]  = b_disperse*len;
	nd[tid] = b_disperse;
	
}


void ConsumerSystem::disperse(float * resource){
	int nt = min(256, nc); int nb = 1; 
	disperse_kernel <<< nb, nt >>> (resource, pos_i_dev, 
									kdsd_dev, RT_dev, 
									cs_dev_XWstates, 
									L, nc, nx,
									lenDisp_dev, nd_dev);	
}


// note: vc_window is as follows (because all quantities are in row arrays):
//		c1 c2 c3 c4 ....
//	t1   
//	t2
//	t3
//	...
//	tw	
//

__global__ void calc_payoffs_kernel(float * rc, float * lend, float * hc, int nc, 
									float * vc_window, float * vc, int tw, int t,
									float b, float cdisp, float charv ){

	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nc) return;

	float v = b*rc[tid] - cdisp*lend[tid] - charv*hc[tid]*hc[tid]; 
	vc_window[ix2(tid, t%tw, nc)] = v;

	float vavg = 0;
	for (int i=0; i<tw; ++i) vavg = vc_window[ix2(tid, i, nc)];
	vavg = vavg/tw;
	
	vc[tid] = vavg;
									
} 

void ConsumerSystem::calcPayoffs(int t){
	int nt = min(256, nc); int nb = 1; 
	calc_payoffs_kernel <<<nb, nt >>> (rc_dev, lenDisp_dev, h_dev, nc, 
									   vc_window_dev, vc_dev, vc_Tw, t,
									   0.002, 0.1, 0.08);
									   
	hipMemcpy2D(&consumers[0].rc, sizeof(Consumer), rc_dev, sizeof(float), sizeof(float), nc, hipMemcpyDeviceToHost);
	hipMemcpy2D(&consumers[0].h, sizeof(Consumer), h_dev, sizeof(float), sizeof(float), nc, hipMemcpyDeviceToHost);
	hipMemcpy2D(&consumers[0].ld, sizeof(Consumer), lenDisp_dev, sizeof(float), sizeof(float), nc, hipMemcpyDeviceToHost);
	hipMemcpy2D(&consumers[0].nd, sizeof(Consumer), nd_dev, sizeof(float), sizeof(float), nc, hipMemcpyDeviceToHost);
	hipMemcpy2D(&consumers[0].vc, sizeof(Consumer), vc_dev, sizeof(float), sizeof(float), nc, hipMemcpyDeviceToHost);
	for (int i=0; i<nc; ++i){
		
	}
									   
}


